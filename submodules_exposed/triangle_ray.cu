#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>

#include "triangle_ray.h"
#include "helpers.h"

#include "vec_math.h"

extern "C" {
__constant__ Params params;
}


static __forceinline__ __device__ void setPayload( float3 p )
{
    optixSetPayload_0( float_as_int( p.x ) );
    optixSetPayload_1( float_as_int( p.y ) );
    optixSetPayload_2( float_as_int( p.z ) );
}

extern "C" __global__ void __raygen__rg()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    float4 ro4 = params.ray_origin[idx.y * params.image_width + idx.x];
    float4 rd4 = params.ray_direction[idx.y * params.image_width + idx.x];

    float3 ray_origin    = make_float3(ro4.x, ro4.y, ro4.z);
    float3 ray_direction = make_float3(rd4.x, rd4.y, rd4.z);
    ray_direction = normalize(ray_direction);
    float3 result = make_float3(0);
    unsigned int p0, p1, p2;
    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        1e-4f,
        100.0f,
        0.0f,
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_NONE,
        0,
        1,
        0,
        p0, p1, p2);

    result.x = int_as_float(p0);
    result.y = int_as_float(p1);
    result.z = int_as_float(p2);

    // Flip Y so that row 0 = bottom (DPG convention)
    // int iy_flipped = params.image_height - 1 - idx.y;
    params.image[idx.y * params.image_width + idx.x] = make_color(result);
}


extern "C" __global__ void __miss__ms()
{
    MissData* miss_data  = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    setPayload(  miss_data->bg_color );
}


extern "C" __global__ void __closesthit__ch()
{
    // When built-in triangle intersection is used, a number of fundamental
    // attributes are provided by the OptiX API, indlucing barycentric coordinates.
    const float2 barycentrics = optixGetTriangleBarycentrics();

    setPayload( make_float3( barycentrics, 1.0f ) );
}
